#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>
#include<stdio.h>
#include <string.h>
#include <iomanip>
#include <fstream>
#include <stack>
#include<sstream>
#include<math.h>
using namespace std;

long long remaining_N2(int , int ,long long );
long long remaining_N(int , int ,int );
void preprocessing(float * , int ,int );
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void ker(float * cormat, float * upper,int n1,int n)
{
long idx = blockDim.x*blockIdx.x+threadIdx.x;
long i = idx%n1;
long j = idx/n1;
if(i<j && i<n1 && j<n)
{
        long tmp=i;
        tmp*=(i+1);
        tmp/=2;
        long tmp_2=i;
        tmp_2*=n;
        tmp_2=tmp_2-tmp;
       tmp_2+=j;
       tmp_2-=i;


upper[tmp_2-1]=cormat[j*n+i];
}
}


__global__ void ker2(float * cormat, float * upper,int n1,int n,long long upper_size,int N,int i_so_far,long long M1)
{
long long idx = blockDim.x;
idx*=blockIdx.x;
idx+=threadIdx.x;
long i = idx/n;
long j = idx%n;

if(i<j && i<n1 && j<n)// &&i<N &&j<N && idx<(n1*n))
{
        long long tmp=i;
        tmp*=(i+1);
        tmp/=2;
        long long tmp_2=i;
        tmp_2*=n;
        tmp_2=tmp_2-tmp;
        tmp_2+=j;
        tmp_2-=i;
        long long indexi=n1;
        indexi*=j;
        indexi=indexi+i;
        upper[tmp_2-1]=cormat[indexi];
//if((i==39001 &&j == 69999)||(i==1 && j==2))
 // printf("\n\n\n thread:  %f ",upper[tmp_2-1]," ",cormat[indexi]);
}

}


int CorMat_2(float* upper_tri, float * BOLD, int N, int L)
{
    long long M1 = (N-1); //computing the  size of correlaion matrix
    M1 *= N;
    M1 /= 2;
    long long total=N*N;//size of total correlation matrix

   // float * total_cormat = new float [total];

    preprocessing( BOLD,  N, L);//Preprocessing fMRI data in CPU
    
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle) ;
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        cout<<"Error in creating cublas handle";
        return stat;
    }
    
    
    float * devBOLD; //Allocating space in GPU for storing fMRI data
    cudaStat = hipMalloc ((void**)&devBOLD, sizeof(float) * L * N) ;
    
    if (cudaStat != hipSuccess)
    {
        cout<<"Error in Cuda Malloc";
        return cudaStat;
    }
    
    stat = hipblasSetMatrix(N, L, sizeof(float), BOLD, N, devBOLD, N);//Copying fMRI data from CPU to GPU
    if (stat != HIPBLAS_STATUS_SUCCESS)
     cout<<"Error in copying data to GPU";
    
    
    const float alpha = 1.0;
    const float beta = 0.0;
    
    float* devCormat;//allocating space in GPU for whole correlation matrix
    hipMalloc ( (void**)&devCormat, sizeof(float) * total) ;
    
    stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, N,N,L,  &alpha, devBOLD, L, devBOLD, L, &beta, devCormat, N);//Performing matrix multiplication (fMRI data to its transpose)
    hipDeviceSynchronize();
    
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        cout<<"Error performing multiplication";
        return stat;
    }
    
    float* dev_upper;//Allocating space for extracting upper triangle part
    hipMalloc ( (void**)&dev_upper, sizeof(float) * M1) ;
    
    int block_size=1024;//number of threads
    long long grid_size=1+((total-1)/block_size);//number of blocks
    ker<<<grid_size,block_size>>>(devCormat,dev_upper,N,N);//performing kernel for extracting and reordering correlations from upper triangle
 memset((void*)upper_tri, 0, sizeof(float) *M1);
    gpuErrchk( hipPeekAtLastError() );
    
    hipMemcpy(upper_tri, dev_upper, sizeof(float) *M1, hipMemcpyDeviceToHost);//copying upper triangle correlation matrix data back to CPU
    
    if (cudaStat != hipSuccess)
    {
        cout<<"Error in Cuda memcpy back to cpu";
        return cudaStat;
    }
    
    hipFree (devCormat);
    hipFree (dev_upper);
    stat = hipblasDestroy(handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        cout<<"Error in destroy";
        return stat;
    }
    return 1;
    
    
}


////////////////////////////////////////

int CorMat_3(float* upper_tri, float * BOLD, int N, int L,long long OOO)
{
    //clock_t first,second;    
    size_t free;int ii=0;
    size_t total_mem;
    hipMemGetInfo(&free,&total_mem);
    long long available_mem = free;
    available_mem/=sizeof(float);
    available_mem-=(N*L);//Getting available memory without
    
    int flag=1;

    preprocessing( BOLD,  N, L);//Preprocessing fMRI data

    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    long long upper_size=(N-1);//computinf size of total correlation matrix
    upper_size*=N;
    upper_size/=2;
    
    float * devBOLD;//initializing normalized fMRI data in GPU
    cudaStat = hipMalloc ((void**)&devBOLD, sizeof(float) * L * N);

    stat = hipblasSetMatrix(N, L, sizeof(float), BOLD, N, devBOLD, N);

    hipblasCreate(&handle) ;
    
    const float alpha = 1.0;
    const float beta = 0.0;
    
    int block,N_prime;
    block=OOO;
    N_prime=N;
    
    float* add_uper_cpu=upper_tri;
    long long M1,temp,temp2=0,temp3=0;
    int so_far=0;
    int pak=0;
    float* devCormat;
    float* dev_upper;
    int ffl=0;
    long long old_cormat_fullsize;
    long long old_M1;
    long long cormat_fullsize;
    while(flag==1)
    {
        cout<<"this is block: "<<block<<"\n\n";        
        if(block==N_prime)//checking for the last chunk
           flag=0;
        
        temp = block;
        temp *= (block +1);
        temp /= 2;
        M1=N_prime;
        M1*=block;
        M1-=temp; //M1 is the size of upper triangle part of chunk


		if(pak!=0)
			{
			hipFree (dev_upper);
			hipFree (devCormat);

			}
            cormat_fullsize=block;
            cormat_fullsize*=N_prime;

            cudaStat=hipMalloc ( (void**)&devCormat, sizeof(float) * cormat_fullsize) ;
            
            if (cudaStat != hipSuccess)
            
            {
                cout<<"Error in Cuda Malloc and status is devcormat: "<<cudaStat;
                return cudaStat;
            }
            
            cudaStat =  hipMalloc ( (void**)&dev_upper, sizeof(float) * M1) ;
            if (cudaStat != hipSuccess)
            
            {
                cout<<"Error in Cuda Malloc and status is devcormat: "<<cudaStat;
                return cudaStat;
            }
  
            
            cout<<"\n IN PAK  0: "<<cormat_fullsize<<" " <<M1<<"*****";
            old_cormat_fullsize=cormat_fullsize;
            old_M1=M1;
            pak++;

        stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, block,N_prime,L,  &alpha, devBOLD+(so_far*L), L, devBOLD+(so_far*L), L, &beta, devCormat, block);//multiply block x L to L x N_prime = block x N_prime

        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            cout<<"error in cublasSgemm, stat is: \n";
            cout<<stat<<"\n";
            return stat;
        }

        hipDeviceSynchronize();

        temp2=block;
        temp2*=N_prime;

        int block_size=1024;
        long long grid_size=1+((temp2-1)/block_size);

        ker2<<<grid_size,block_size>>>(devCormat,dev_upper,block,N_prime,upper_size,N,ii,M1);
        
	memset((void*)add_uper_cpu, 0, sizeof(float) *M1); 
	
        hipDeviceSynchronize();
        ii+=block;
        
        gpuErrchk( hipPeekAtLastError() );
        
        cudaStat= hipMemcpy(add_uper_cpu, dev_upper, sizeof(float) *M1, hipMemcpyDeviceToHost);

        if (cudaStat != hipSuccess)
        {
            cout<<"cudamalloc add_uper: \n";
            cout<<stat<<"\n";
            return stat;
        }

        temp3+=M1;
        add_uper_cpu=upper_tri+temp3;
        so_far+=block;
        
        if(N_prime>block)
        {
            N_prime=N_prime-block;
            block=remaining_N2( N_prime, L, available_mem);
          
            if(N_prime  <block)//checking last chunk
             block=N_prime;

        }
    }
    hipFree (devBOLD);
    hipFree (dev_upper);
    hipFree (devCormat);

    stat = hipblasDestroy(handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        cout<<"error in destroy";
        return stat;
    }
    
    
    return 1;
}





///////////////////////////////////////

